#include "hip/hip_runtime.h"
#include "testkernel.h"

// KERNEL
__global__ void testKernel(float *out, float *a, float *b, int n){
  for(int i=0; i<n; i++)
  {
    out[i] = a[i] + b[i];
  }
}

// CALLER
void kernelCaller(float *out, float *a, float *b, int n){

  float *da, *db, *dout;

  // Dedicate memory on device
  hipMalloc((void **) &da, sizeof(float) * n);
  hipMalloc((void **) &db, sizeof(float) * n);
  hipMalloc((void **) &dout, sizeof(float) * n);

  // Copy host to device
  hipMemcpy(da, a, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(db, b, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(dout, out, sizeof(float) * n, hipMemcpyHostToDevice);

  // Execute kernel
  testKernel<<<1, 1>>>(dout, da, db, n);

  // Copy device to host
  hipMemcpy(a, da, sizeof(float) * n, hipMemcpyDeviceToHost);
  hipMemcpy(b, db, sizeof(float) * n, hipMemcpyDeviceToHost);
  hipMemcpy(out, dout, sizeof(float) * n, hipMemcpyDeviceToHost);
}
