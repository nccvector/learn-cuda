
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define N 1000

// KERNEL
__global__ void testKernel(float *out, float *a, float *b, int n){
  for(int i=0; i<N; i++)
  {
    out[i] = a[i] + b[i];
  }
}

// CALLER
void kernelCaller(float *out, float *a, float *b, int n){

  float *da, *db, *dout;

  // Dedicate memory on device
  hipMalloc((void **) &da, sizeof(float) * N);
  hipMalloc((void **) &db, sizeof(float) * N);
  hipMalloc((void **) &dout, sizeof(float) * N);

  // Copy host to device
  hipMemcpy(da, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(db, b, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(dout, out, sizeof(float) * N, hipMemcpyHostToDevice);

  // Execute kernel
  testKernel<<<1, 1>>>(dout, da, db, N);

  // Copy device to host
  hipMemcpy(a, da, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipMemcpy(b, db, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipMemcpy(out, dout, sizeof(float) * N, hipMemcpyDeviceToHost);
}

int main() {
  // Create some memory in host
  std::vector<float> a, b, out;
  a.reserve(N);
  b.reserve(N);
  out.reserve(N);

  for (int i = 0; i < N; i++) {
    a[i] = 2;
    b[i] = 3;
    out[i] = 0;
  }

  kernelCaller(out.data(), a.data(), b.data(), N);

  for (int i=0; i<N; i++)
  {
    std::cout << out[i] << std::endl;
  }
}
