
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define N 1000

// KERNEL
__global__ void testKernel(float *out, float *a, float *b, int n){
  for(int i=0; i<n; i++)
  {
    out[i] = a[i] + b[i];
  }
}

// CALLER
void kernelCaller(float *out, float *a, float *b, int n){

  float *da, *db, *dout;

  // Dedicate memory on device
  hipMalloc((void **) &da, sizeof(float) * n);
  hipMalloc((void **) &db, sizeof(float) * n);
  hipMalloc((void **) &dout, sizeof(float) * n);
  // Copy host to device
  hipMemcpy(da, a, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(db, b, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(dout, out, sizeof(float) * n, hipMemcpyHostToDevice);

  // Execute kernel
  testKernel<<<1, 1>>>(dout, da, db, n);

  // Copy device to host
  hipMemcpy(a, da, sizeof(float) * n, hipMemcpyDeviceToHost);
  hipMemcpy(b, db, sizeof(float) * n, hipMemcpyDeviceToHost);
  hipMemcpy(out, dout, sizeof(float) * n, hipMemcpyDeviceToHost);
}

int main() {
  // Create some memory in host
  std::vector<float> a, b, out;
  a.reserve(N);
  b.reserve(N);
  out.reserve(N);

  for (int i = 0; i < N; i++) {
    a[i] = 2;
    b[i] = 3;
    out[i] = 0;
  }

  kernelCaller(out.data(), a.data(), b.data(), N);

  for (int i=0; i<N; i++)
  {
    std::cout << out[i] << std::endl;
  }
}
