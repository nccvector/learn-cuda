#include "hip/hip_runtime.h"
#include "testkernel.cuh"

// KERNEL
__global__ void testKernel(float *out, float *a, float *b, int N){
  for(int i=0; i<N; i++)
  {
    out[i] = a[i] + b[i];
  }
}

// CALLER
void kernelCaller(float *out, float *a, float *b, int N){

  float *da, *db, *dout;

  // Dedicate memory on device
  hipMalloc((void **) &da, sizeof(float) * N);
  hipMalloc((void **) &db, sizeof(float) * N);
  hipMalloc((void **) &dout, sizeof(float) * N);

  // Copy host to device
  hipMemcpy(da, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(db, b, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(dout, out, sizeof(float) * N, hipMemcpyHostToDevice);

  // Execute kernel
  testKernel<<<1, 1>>>(dout, da, db, N);

  // Copy device to host
  hipMemcpy(a, da, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipMemcpy(b, db, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipMemcpy(out, dout, sizeof(float) * N, hipMemcpyDeviceToHost);
}